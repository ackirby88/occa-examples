#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void reverseKernel(int N, float *d_a){
	   
  // Convert thread and thread-block indices into array index 
  const int n  = threadIdx.x + blockDim.x*blockIdx.x;

  // If index is in [0,N/2], swap values
  if(n<N/2){
    float an = d_a[n];
    d_a[n] = d_a[N-1-n];
    d_a[N-1-n] = an;
  }
}

int main(int argc,char **argv){
  int N = 31; // size of array for this DEMO
    
  float *d_a;  // Allocate DEVICE array
  hipMalloc(&d_a, N*sizeof(float));
    
  int nThreadsPerBlock = 10;
  int nThreadBlocks = ((N/2)+nThreadsPerBlock-1)/nThreadsPerBlock;
  
  // HOST array
  float *h_a = (float*) malloc(N*sizeof(float));
  for(int i = 0; i < N; i++){
    h_a[i] = i;
    printf("h_a[%d]=%f\n",i,h_a[i]);
  }
    
  // Copy data from HOST to DEVICE
  hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
   
  // Queue kernel on DEVICE
  reverseKernel <<< nThreadBlocks, nThreadsPerBlock >>> (N, d_a);
    
  // Transfer result from DEVICE to HOST
  hipMemcpy(h_a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);
    
  // Print out result
  for(int n=0;n<N;++n){
    printf("h_a[%d] = %f\n", n, h_a[n]);
  }

  free(h_a);
  return 0;
}
